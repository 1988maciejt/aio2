#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <algorithm>
#include <vector>
#include <bitset>
    
`cpu_debug = kwargs.get('cpu_debug', False)
`scan_length = kwargs.get('scan_length', 16)
`scan_count = kwargs.get('scan_count', 8)
`cuda_blocks = kwargs.get('cuda_blocks', 64)
`cuda_threads = kwargs.get('cuda_threads', 64)
`cuda_grid_size = cuda_blocks * cuda_threads
`compactor_registers = kwargs.get('compactor_registers', [])
`global_xor_exists = ([0, 0] in compactor_registers)
`compactor_registers_count = len(kwargs.get('compactor_registers', []))
`compactor_register_size = kwargs.get('scan_length', 16)+kwargs.get('scan_count', 8)
`compactor_register_words = (compactor_register_size-1) // 16 + 1
`compactor_given_result = kwargs.get('compactor_value', None)
`if compactor_given_result is None:
`   Aio.print("Compactor value not given!!!!")
`endif
`shift_register_size = scan_count
`shift_register_words = (shift_register_size-1) // 16 + 1
    
// max tree branches for host/dev:  200000000
#define TREE_BRANCHES               `(kwargs.get('max_tree_branches', 20000000)`) 
#define MAX_LUT_SIZE                `(kwargs.get('max_lut_size', 4000000)`) 

#define SCAN_LENGTH                 `(scan_length`)
#define SCAN_REGISTER_WORDS         `((scan_length-1) // 16 + 1`)
#define SCAN_COUNT                  `(scan_count`)

#define COMPACTOR_REGISTERS_COUNT   `(compactor_registers_count`)
#define COMPACTOR_REGISTER_SIZE     `(compactor_register_size`)
#define COMPACTOR_REGISTER_WORDS    `(compactor_register_words`)

#define MAX_TOTAL_FAIL_COUNT        `(kwargs.get('max_total_fails', 8)`)
#define FAILS_PER_CLOCK_CYCLE       `(kwargs.get('max_fails_per_clock_cycle', 3)`)
#define FAILS_HORIZONTAL_DISTANCE   `(kwargs.get('max_fails_horizontal_distance', 5)`)
#define FAILS_VERTICAL_DISTANCE     `(kwargs.get('max_fails_vertical_distance', 5)`)
    
using namespace std;


    
struct TreeItem {
    short ScanMinIndex = -1;
    short ScanMaxIndex = -1;
    short FailMap[SCAN_LENGTH];
};

struct LUTRow {
    short First = -1;
    short Last = -1;
    short FailCount = 0;
    short LUT[FAILS_PER_CLOCK_CYCLE];
};

// COMPACTOR REGISTERS
struct CompactorRegister {
    unsigned short Reg[COMPACTOR_REGISTER_WORDS];
};
struct CompactorRegisters {
    short Length = 0;
    CompactorRegister Regs[COMPACTOR_REGISTERS_COUNT];
};
void printCompactorRegisters(CompactorRegisters* Regs) {
    for (int i = 0; i < COMPACTOR_REGISTERS_COUNT; i++) {
        cout << " - REG " << i << ":\\t";
        for (int j = COMPACTOR_REGISTER_WORDS-1; j >= 0; j--) {
            cout << bitset<16>(Regs->Regs[i].Reg[j]);
            if (j > 0) cout << "";
        }
        cout << "      ";
        for (int j = COMPACTOR_REGISTER_WORDS-1; j >= 0; j--) {
            cout << (Regs->Regs[i].Reg[j]);
            if (j > 0) cout << ",";
        }
        cout << endl;
    }
}
__host__ __device__
short getCompactorBit(CompactorRegisters* Regs, short& RegIndex, short& BitIndex) {
    if ((Regs->Regs[RegIndex].Reg[BitIndex / 16]) & (1 << (BitIndex % 16))) return 1;
    return 0;
}
__host__ __device__
void setCompactorBit(CompactorRegisters* Regs, short& RegIndex, short& BitIndex, short BitValue) {
    if (BitValue) {
        Regs->Regs[RegIndex].Reg[BitIndex / 16] |= (1 << (BitIndex % 16));
    } else {
        Regs->Regs[RegIndex].Reg[BitIndex / 16] &= ~(1 << (BitIndex % 16));
    }
}
__host__ __device__
void pushCompactorValue(CompactorRegisters* Regs, short* BitValues) {
    for (short ri = 0; ri < COMPACTOR_REGISTERS_COUNT; ri++) {
        setCompactorBit(Regs, ri, Regs->Length, BitValues[ri]);
    }
    Regs->Length++;
    if (Regs->Length > COMPACTOR_REGISTER_SIZE)
        Regs->Length--;
}
__host__ __device__
void clearCompactorRegisters(CompactorRegisters* Regs) {
    Regs->Length = 0;
    for (short ri = 0; ri < COMPACTOR_REGISTERS_COUNT; ri++) {
        for (short i = 0; i < COMPACTOR_REGISTER_WORDS; i++) {
            Regs->Regs[ri].Reg[i] = 0;
        }
    }
}
__host__ __device__
bool areCompactorsEqual(CompactorRegisters* SimComp, CompactorRegisters* Reference) {
    for (short cindex = 0; cindex < SimComp->Length; cindex++) {
        for (short ri = 0; ri < COMPACTOR_REGISTERS_COUNT; ri++) {
            if (getCompactorBit(SimComp, ri, cindex) != getCompactorBit(Reference, ri, cindex)) {
                return false;
            }
        }
    }
    return true;
}
    
// SHIFT REGISTER
struct ShiftRegister {
    unsigned short Reg[`(shift_register_words`)];
};
__host__ __device__ 
void setShiftRegisterBit(ShiftRegister* Reg, short& BitIndex, short& BitValue) {
    if (BitValue) {
        Reg->Reg[BitIndex / 16] |= (1 << (BitIndex % 16));
    } else {
        Reg->Reg[BitIndex / 16] &= ~(1 << (BitIndex % 16));
    }
}
__host__ __device__
short getShiftRegisterBit(ShiftRegister* Reg, short BitIndex) {
    if ((Reg->Reg[BitIndex / 16]) & (1 << (BitIndex % 16))) return 1;
    return 0;
}
__host__ __device__
void shiftShiftRegister(ShiftRegister* Reg) {
    for (short i =  `(shift_register_words-1`); i >= 0; i--) {
        Reg->Reg[i] <<= 1;
        if ((i > 0) && (Reg->Reg[i-1] & (1<<15))) {
            Reg->Reg[i] |= 1;
        }
    }
}
__host__ __device__
void clearShiftRegister(ShiftRegister* Reg) {
    for (short i = 0; i < `(shift_register_words`); i++) {
        Reg->Reg[i] = 0;
    }
}
__host__ __device__
void flipShiftRegisterBit(ShiftRegister* Reg, short& BitIndex) {
    Reg->Reg[BitIndex / 16] ^= (1 << (BitIndex % 16));
}
void printShiftRegister(ShiftRegister* Reg) {
    for (int j = `(shift_register_words`)-1; j >= 0; j--) {
        cout << bitset<16>(Reg->Reg[j]);
        if (j > 0) cout << "";
    }
    cout << "      ";
    for (int j = `(shift_register_words`)-1; j >= 0; j--) {
        cout << (Reg->Reg[j]);
        if (j > 0) cout << ",";
    }
    cout << endl;
}
    
// SEARCHING TREE
__host__ __device__
TreeItem getCopiedTreeItem(TreeItem* MyTree, int Index) {
    TreeItem NewItem;
    NewItem.ScanMinIndex = MyTree[Index].ScanMinIndex;
    NewItem.ScanMaxIndex = MyTree[Index].ScanMaxIndex;
    for (int i = 0; i < SCAN_LENGTH; i++) {
        NewItem.FailMap[i] = MyTree[Index].FailMap[i];
    }
    return NewItem;
}
__host__ __device__
TreeItem getCopiedTreeItem(TreeItem* TItem) {
    TreeItem NewItem;
    NewItem.ScanMinIndex = TItem->ScanMinIndex;
    NewItem.ScanMaxIndex = TItem->ScanMaxIndex;
    for (int i = 0; i < SCAN_LENGTH; i++) {
        NewItem.FailMap[i] = TItem->FailMap[i];
    }
    return NewItem;
}
__host__ __device__
void addFailToTreeBranch(TreeItem* TreeBranch, int FailIndex, int& LutIndex, LUTRow* Lut) {
    TreeBranch->FailMap[FailIndex] = LutIndex;
    if (Lut[LutIndex].FailCount > 0) {
        if (TreeBranch->ScanMinIndex < 0 || Lut[LutIndex].First < TreeBranch->ScanMinIndex) {
            TreeBranch->ScanMinIndex = Lut[LutIndex].First;
        }
        if (Lut[LutIndex].Last > TreeBranch->ScanMaxIndex) {
            TreeBranch->ScanMaxIndex = Lut[LutIndex].Last;
        }
    }
}
__host__ __device__
void copyTreeItem(TreeItem* MyTree, int Src, int Dst) {
    MyTree[Dst].ScanMinIndex = MyTree[Src].ScanMinIndex;
    MyTree[Dst].ScanMaxIndex = MyTree[Src].ScanMaxIndex;
    for (int i = 0; i < SCAN_LENGTH; i++) {
        MyTree[Dst].FailMap[i] = MyTree[Src].FailMap[i];
    }
}
__host__ __device__
void copyTreeItem(TreeItem* MyTree, TreeItem* Src, int Dst) {
    MyTree[Dst].ScanMinIndex = Src->ScanMinIndex;
    MyTree[Dst].ScanMaxIndex = Src->ScanMaxIndex;
    for (int i = 0; i < SCAN_LENGTH; i++) {
        MyTree[Dst].FailMap[i] = Src->FailMap[i];
    }
}
__host__ __device__
void disableTreeBranch(TreeItem* MyTree, int& Index) {
    MyTree[Index].ScanMinIndex = -2;
}
__host__ __device__
void cleanTree(TreeItem* MyTree, int* TreeSize) {
    bool DidSomething = true;
    while (DidSomething) {
        int i = 0;
        DidSomething = false;
        while (i < *TreeSize) {
            if (MyTree[i].ScanMinIndex < -1) {
                copyTreeItem(MyTree, (*TreeSize)-1, i);
                (*TreeSize)--;
                DidSomething = true;
            }
            i++;
        }
    }
}
void printTreeItem(TreeItem& MyTree) {
    cout << "TreeItem >> ";
    cout << "Min: " << MyTree.ScanMinIndex << " ";
    cout << "Max: " << MyTree.ScanMaxIndex << " ";
    cout << "Fails: ";
    for (int i = 0; i < SCAN_LENGTH; i++) {
        cout << MyTree.FailMap[i] << " ";
    }
    cout << endl;
}
    
// LUT
__host__ __device__
void addToLUT(LUTRow* Row, short Value) {
    if (Row->FailCount < FAILS_PER_CLOCK_CYCLE) {
        if ((Row->First < 0) || (Value < Row->First)) Row->First = Value;
        if ((Row->Last < 0) || (Value > Row->Last)) Row->Last = Value;
        Row->LUT[Row->FailCount] = Value;
        Row->FailCount++;
    }
}
void getLUT(LUTRow* Lut, int& LutSize) {
    int LutMax = LutSize;
    LUTRow* Row = new LUTRow;
    Row->First = -1;
    Row->FailCount = 0;
    Lut[LutSize] = *Row;
    LutSize = 1;    
    int kmax = FAILS_PER_CLOCK_CYCLE;
    if (kmax > SCAN_COUNT) kmax = SCAN_COUNT;
    for (int k = 1; k <= kmax; k++) {
        vector<bool> v(SCAN_COUNT);
        fill(v.end() - k, v.end(), true);
        do {
            short First = -1;
            short Last = -1;
            Row = new LUTRow;
            for (short i = 0; i < SCAN_COUNT; ++i) {
                if (v[i]) {
                    Last = (i);
                    if (First == -1) First = Last;
                    addToLUT(Row, Last);
                }
            }
            if ((Last - First) <= FAILS_VERTICAL_DISTANCE) {
                Lut[LutSize] = *Row;
                LutSize++;
                if (LutSize >= LutMax) {
                    cout << "LUT size exceeded" << endl;
                    return;
                }
            } else {
                delete Row;
            }
        } while (next_permutation(v.begin(), v.end()));
    }
}
void printLutRow(LUTRow& LR) {
    cout << "LUTRow >> ";
    cout << "First: " << LR.First << " ";
    cout << "Last: " << LR.Last << " ";
    cout << "Fails: ";
    for (int i = 0; i < LR.FailCount; i++) {
        cout << LR.LUT[i] << " ";
    }
    cout << endl;
}
    
`if not cpu_debug:
__global__ 
`endif
void kernel(TreeItem* MyTree, int TreeSize, int* NewTreeSize, LUTRow* Lut, int LutSize,
        CompactorRegisters* GivenCompactorOutput, CompactorRegisters* SimulatorCompactor, int Cycles) {
`if cpu_debug:
    int EnvIndex = 0;
`else:
    int EnvIndex = threadIdx.x + blockIdx.x * blockDim.x;
`endif
    int BranchIndex = EnvIndex;
`for rdi in range(len(compactor_registers)):
`   reg_definition = compactor_registers[rdi]
`   if reg_definition[0] != 0:
    short aux;
`       break
`   endif
`endfor
    short OutputValues[COMPACTOR_REGISTERS_COUNT];
`for rdi in range(len(compactor_registers)):
`   reg_definition = compactor_registers[rdi]
`   if reg_definition[0] != 0:
    ShiftRegister ShiftReg`(rdi`);
`   endif
`endfor
    while (BranchIndex < TreeSize) {
        bool BranchOverwritten = false;
        TreeItem BaseBranch = getCopiedTreeItem(MyTree, BranchIndex);
        CompactorRegisters MyCompactor = SimulatorCompactor[EnvIndex];
        for (int NewLutPosition = 0; NewLutPosition < LutSize; NewLutPosition++) {
            TreeItem MyBranch = getCopiedTreeItem(&BaseBranch);
            if (Cycles <= SCAN_LENGTH)
                addFailToTreeBranch(&MyBranch, Cycles-1, NewLutPosition, Lut);
            short TotalFails = 0;
            short FirstNonZeroCycle = -1;
            short LastNonZeroCycle = -1;
            for (short CycleIndex = 0; CycleIndex < (Cycles>SCAN_LENGTH ? SCAN_LENGTH : Cycles); CycleIndex++) {
                short fc = Lut[MyBranch.FailMap[CycleIndex]].FailCount;
                if (fc > 0) {
                    LastNonZeroCycle = CycleIndex;
                    if (FirstNonZeroCycle < 0) {
                        FirstNonZeroCycle = CycleIndex;
                    }
                }
                TotalFails += fc;
            }
            if (TotalFails > MAX_TOTAL_FAIL_COUNT) {
                continue;
            } 
            if (MyBranch.ScanMaxIndex - MyBranch.ScanMinIndex > FAILS_VERTICAL_DISTANCE) {
                continue;
            }
            if (LastNonZeroCycle - FirstNonZeroCycle > FAILS_HORIZONTAL_DISTANCE) {
                continue;
            }
            for (short i = 0; i < COMPACTOR_REGISTERS_COUNT; i++) {
                OutputValues[i] = 0;
            }
            clearCompactorRegisters(&MyCompactor);
`for rdi in range(len(compactor_registers)):
`   reg_definition = compactor_registers[rdi]
`   if reg_definition[0] != 0:
            clearShiftRegister(&ShiftReg`(rdi`));
`   endif
`endfor
`if cpu_debug:
            if (Cycles > 32) {
                cout << "BRANCH:";
                printTreeItem(MyBranch);
            }
`endif
            for (short CycleIndex = 0; CycleIndex < Cycles; CycleIndex++) {
                short CycleScanValues[SCAN_COUNT];
                for (short i = 0; i < SCAN_COUNT; i++) {
                    CycleScanValues[i] = 0;
                }            
                if (CycleIndex <= SCAN_LENGTH) {
                    LUTRow Fails = Lut[MyBranch.FailMap[CycleIndex]];
                    for (short fi = 0; fi < Fails.FailCount; fi++) {
                        CycleScanValues[Fails.LUT[fi]] = 1;
                    }
                }
`for rdi in range(len(compactor_registers)):
`   reg_definition = compactor_registers[rdi]
`   if reg_definition[0] == 0:
                OutputValues[`(rdi`)] = 0;
                for (short i = 0; i < SCAN_COUNT; i++) {
                    OutputValues[`(rdi`)] ^= CycleScanValues[i];
                }
`   else:
                OutputValues[`(rdi`)] = getShiftRegisterBit(&ShiftReg`(rdi`), `(shift_register_size-1`));
                shiftShiftRegister(&ShiftReg`(rdi`));
`       dir_up = 1 if reg_definition[0] > 0 else false
`       xoring = abs(reg_definition[0])
`       offset = reg_definition[1]
                for (short SCIndex = 0; SCIndex < SCAN_COUNT; SCIndex++) {
`       if xoring == 1:
`           if dir_up:
                    aux = CycleScanValues[(SCIndex + `(offset`)) % SCAN_COUNT];
`           else:
                    aux = CycleScanValues[((SCAN_COUNT-1) - (SCIndex + `(offset`))) % SCAN_COUNT];
`           endif
`       else:
                    aux = 0;
                    for (short k = 0; k < `(xoring`); k++) {
`           if dir_up:
                        aux ^= CycleScanValues[(SCIndex + `(offset`) + k) % SCAN_COUNT];
`           else:
                        aux ^= CycleScanValues[((SCAN_COUNT-1) - (SCIndex + `(offset`) + k)) % SCAN_COUNT];
`           endif
                    }
`       endif
                    if (aux) flipShiftRegisterBit(&ShiftReg`(rdi`), SCIndex);
                }
`   endif
`endfor
`if cpu_debug:
                if (Cycles > 32) {
                    cout << "Cycle " << CycleIndex << " : OutpuCycleScanValuestValues = ";
                    for (short i = 0; i < SCAN_COUNT; i++) {
                        cout << CycleScanValues[i];
                    }            
                    cout << "   OutputVal = ";
                    for (short i = 0; i < COMPACTOR_REGISTERS_COUNT; i++) {
                        cout << OutputValues[i] ;
                    }
                    cout << endl;
                }
`endif
                pushCompactorValue(&MyCompactor, OutputValues);
            }
            if (areCompactorsEqual(&MyCompactor, GivenCompactorOutput)) {
                int AuxIndex;
                if (BranchOverwritten) {
`if cpu_debug:
                    AuxIndex = (*NewTreeSize);
                    (*NewTreeSize)++;
                //cout << "CORRECT BRANCH:" << AuxIndex << " " << *NewTreeSize << endl;
`else:
                    AuxIndex = atomicAdd(NewTreeSize, 1);
`endif
                } else {
                    BranchOverwritten = true;
                    AuxIndex = BranchIndex;
                }
                copyTreeItem(MyTree, &MyBranch, AuxIndex);
            }
`if cpu_debug:
            if (Cycles > 32) {
                printCompactorRegisters(&MyCompactor);
            }
`endif
            if (Cycles > SCAN_LENGTH) {
                break;
            }
        }
        if (!BranchOverwritten) {
            disableTreeBranch(MyTree, BranchIndex);
        }
`if cpu_debug:
        BranchIndex += 1;
`else:
        BranchIndex += blockDim.x * gridDim.x;
`endif
    }
}
    
__global__ 
void kernel_cleanup(TreeItem* MyTree, int* TreeSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid == 0) {
        cleanTree(MyTree, TreeSize);
    }
}
    
int main() {
    clock_t global_c_start = clock();
    clock_t c_start = 0;
    clock_t c_end = 0;
    double time_elapsed_ms = 0;
    
    cout << "['''" << endl;
    cout << "CUDA --------------------------------" << endl;
    cout << "Blocks                      : " << `(cuda_blocks`) << endl;
    cout << "Threads per block           : " << `(cuda_threads`) << endl;

    cout << "COMPACTOR ----------------------------" << endl;
    cout << "# Compactor Registers      : " << COMPACTOR_REGISTERS_COUNT << endl;
    cout << "Compactor Register length  : " << COMPACTOR_REGISTER_SIZE << endl;
    cout << "CompactorRegister size     : " << sizeof(CompactorRegister) << " B" << endl;
    cout << "CompactorRegisters size    : " << sizeof(CompactorRegisters) << " B" << endl;
    CompactorRegisters* SimulatorCompactor = new CompactorRegisters[`(cuda_grid_size`)];
    cout << "SimulatorCompactor size    : " << `(cuda_grid_size`) * sizeof(CompactorRegisters) / (1024) << " kB" << endl;
    CompactorRegisters* SimulatorCompactor_dev = nullptr;
    c_start = clock();
    hipMalloc(&SimulatorCompactor_dev, `(cuda_grid_size`) * sizeof(CompactorRegisters));
    c_end = clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "GPU memory allocated in    : " << time_elapsed_ms << " ms" << endl;
    c_start = clock();
    hipMemcpy(SimulatorCompactor_dev, SimulatorCompactor, `(cuda_grid_size`) * sizeof(CompactorRegisters), hipMemcpyHostToDevice);
    c_end = clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "Compactors copied to GPU in :" << time_elapsed_ms << " ms" << endl;
`if not cpu_debug:
    delete[] SimulatorCompactor;
`endif
    
    // FailMap at compactor output
    CompactorRegisters* GivenCompactorOutput = new CompactorRegisters;
`for c_reg in range(compactor_registers_count):
`    r_val = compactor_given_result[c_reg].copy()
`    for c_word in range(compactor_register_words):
`        result = ''
`        for c_bit in range(16):
`            result = str(r_val[0]) + result
`            r_val <<= 1
`        end
    GivenCompactorOutput->Regs[`(c_reg`)].Reg[`(c_word`)] = 0b`(result`);
`    end
`end
    cout << "Given compactor reg values :" << endl;
    printCompactorRegisters(GivenCompactorOutput);
    CompactorRegisters* GivenCompactorOutput_dev = nullptr;
    hipMalloc(&GivenCompactorOutput_dev, sizeof(CompactorRegisters));
    hipMemcpy(GivenCompactorOutput_dev, GivenCompactorOutput, sizeof(CompactorRegisters), hipMemcpyHostToDevice);

    cout << "LUT ----------------------------------" << endl;
    int LutSize = MAX_LUT_SIZE;
    LUTRow* LutAux = new LUTRow[LutSize];
    getLUT(LutAux, LutSize);
    LUTRow* Lut = (LUTRow*)malloc(LutSize * sizeof(LUTRow));
    memcpy(Lut, LutAux, LutSize * sizeof(LUTRow));
    delete[] LutAux;
    // LUT is ready in Lut[LutSize]
    cout << "# Fails per clock cycle    : " << FAILS_PER_CLOCK_CYCLE << endl;
    cout << "LUT row size               : " << sizeof(LUTRow) << " B" << endl;
    cout << "LUT items                  : " << LutSize << endl;
    cout << "LUT size                   : " << LutSize * sizeof(LUTRow) << " B" << endl;
`if cpu_debug:
    for (int i = 0; i < LutSize; i++) {
        cout << i << "\t: ";
        printLutRow(Lut[i]);
    }
`endif
    LUTRow* Lut_dev = nullptr;
    c_start = clock();
    hipMalloc(&Lut_dev, LutSize * sizeof(LUTRow));
    c_end = clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "GPU memory allocated in    : " << time_elapsed_ms << " ms" << endl;
    c_start = clock();
    hipMemcpy(Lut_dev, Lut, LutSize * sizeof(LUTRow), hipMemcpyHostToDevice);
    c_end = clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "LUT copied to GPU in       : " << time_elapsed_ms << " ms" << endl;
    
    cout << "SEARCHING TREE -----------------------" << endl;
    TreeItem* SearchingTree = new TreeItem[TREE_BRANCHES];
    cout << "TreeItem size              : " << sizeof(TreeItem) << " B" << endl;
    cout << "# Max branches             : " << TREE_BRANCHES / 1000000 << "M" << endl;
    cout << "Max tree size              : " << TREE_BRANCHES * sizeof(TreeItem) / (1024*1024) << " MB" << endl;
    // CUDA memory allocation
    c_start = clock();
    TreeItem* SearchingTree_dev = nullptr;
    hipMalloc(&SearchingTree_dev, TREE_BRANCHES * sizeof(TreeItem));
    int* NewTreeSize_dev = nullptr;
    hipMalloc(&NewTreeSize_dev, sizeof(int));
    c_end = clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "GPU memory allocated in    : " << time_elapsed_ms << " ms" << endl;
    int TreeSize = 0;
    for (int i = 0; i < LutSize; i++) {
        SearchingTree[TreeSize].ScanMinIndex = Lut[i].First;
        SearchingTree[TreeSize].ScanMaxIndex = Lut[i].Last;
        SearchingTree[TreeSize].FailMap[0] = i;
        TreeSize++;
    }
    cout << "Branches before 1st iter.  : " << TreeSize << endl;
    c_start = clock();
    hipMemcpy(SearchingTree_dev, SearchingTree, TreeSize * sizeof(TreeItem), hipMemcpyHostToDevice);
    hipMemcpy(NewTreeSize_dev, &TreeSize, sizeof(int), hipMemcpyHostToDevice);
    c_end = clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "Tree size before 1st iter. : " << TreeSize * sizeof(TreeItem) / (1024) << " kB" << endl;
    cout << "Tree copied to GPU in      : " << time_elapsed_ms << " ms" << endl;
    cout << "--------------------------------------" << endl;
    cout << endl;

    cout << "SIMULATION START =====================" << endl;

    int* NewTreeSize = new int;
    short Cycle = 2;
    while (Cycle <= COMPACTOR_REGISTER_SIZE) {
        cout << "--- Cycle " << Cycle << "\t---" << endl;
        hipMemcpy(SearchingTree_dev, SearchingTree, TreeSize * sizeof(TreeItem), hipMemcpyHostToDevice);
        hipMemcpy(NewTreeSize_dev, &TreeSize, sizeof(int), hipMemcpyHostToDevice);
        c_start = clock();
`if cpu_debug:
        *NewTreeSize = TreeSize;
        kernel(SearchingTree, TreeSize, NewTreeSize, Lut, LutSize,
            GivenCompactorOutput, SimulatorCompactor, Cycle);
        TreeSize = *NewTreeSize;
`else:
        kernel<<<`(cuda_blocks`), `(cuda_threads`)>>>(SearchingTree_dev, TreeSize, NewTreeSize_dev, Lut_dev, LutSize,
            GivenCompactorOutput_dev, SimulatorCompactor_dev, Cycle);
`endif
        c_end = clock();
        time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
        cout << "Kernel execution time      : " << time_elapsed_ms << " ms" << endl;
`if not cpu_debug:
        hipMemcpy(&TreeSize, NewTreeSize_dev, sizeof(int), hipMemcpyDeviceToHost);
`endif
        cout << "Tree branches before clean : " << TreeSize << endl;
        cout << "Tree size before clean     : " << TreeSize * sizeof(TreeItem) / (1024*1024) << " MB" << endl;
`if not cpu_debug:
        hipMemcpy(SearchingTree, SearchingTree_dev, TreeSize * sizeof(TreeItem), hipMemcpyDeviceToHost);
`endif
        //kernel_cleanup<<<1, 1>>>(SearchingTree_dev, NewTreeSize_dev);
        //hipMemcpy(&TreeSize, NewTreeSize_dev, sizeof(int), hipMemcpyDeviceToHost);
        cleanTree(SearchingTree, &TreeSize);
        cout << "Tree branches - final      : " << TreeSize << endl;
        cout << "Tree size - final          : " << TreeSize * sizeof(TreeItem) / (1024*1024) << " MB" << endl;
        if (Cycle ==  SCAN_LENGTH)
            Cycle = COMPACTOR_REGISTER_SIZE;
        else
            Cycle += 1;
    }
    hipMemcpy(SearchingTree, SearchingTree_dev, TreeSize * sizeof(TreeItem), hipMemcpyDeviceToHost);
    
    clock_t global_c_end = clock();
    time_elapsed_ms = 1000.0 * (global_c_end-global_c_start) / CLOCKS_PER_SEC;
    cout << "SIMULATION END =======================" << endl;
    cout << "Total time elapsed          : " << time_elapsed_ms << " ms" << endl;

    cout << "'''," << endl;

    cout << "[ " << endl;
    for (int LutIndex = 0; LutIndex < LutSize; LutIndex++) {
        cout << "[";
        for (short i = 0; i < Lut[LutIndex].FailCount; i++) {
            cout << Lut[LutIndex].LUT[i] << ",";
        }
        cout << "],";
    }
    cout << "]," << endl;

    cout << "[ " << endl;
    for (int TreeIndex = 0; TreeIndex < TreeSize; TreeIndex++) {
        cout << "[";
        for (short i = 0; i < SCAN_LENGTH; i++) {
            cout << SearchingTree[TreeIndex].FailMap[i] << ",";
        }
        cout << "],";
    }
    cout << "]," << endl;

    cout << TreeSize << "," << time_elapsed_ms << "," << endl;

    cout << "]" << endl;
    return 0;
}